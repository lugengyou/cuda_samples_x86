﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 1

#define M 512
#define L 512
#define N 512


/*********************************** utils functions ***********************************/
// 比较结果
bool compare_result(int* a, int* b) {
	for (int i = 0; i < M * N; ++i) {
		if (a[i] != b[i]) {
			printf("Error: %d %d %d\n", i, a[i], b[i]);
			return false;
		}
	}
	return true;
}


/*********************************** c model ***********************************/
// 朴素矩阵乘法
void matrix_mul_c(int* a, int* b, int* c) {

    for(int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            int sum = 0;
            for (int k = 0; k < L; ++k) {
                sum += a[i * L + k] * b[k * N + j];
            }
            c[i * N + j] = sum;
        }
    }
}

// 循环交换，a 的 一个元素与 b 的一行元素相成
void matrix_mul_c_v2(int* a, int* b, int* c) {
    // set c to 0
    for (int i = 0; i < M * N; ++i) {
        c[i] = 0;
    }

    for (int i = 0; i < M; ++i) {
        // 交换循环
        for (int k = 0; k < L; ++k) {
            int tmp = a[i * L + k]; // 取 a 的一个元素
            for (int j = 0; j < N; ++j) { // 与 b 的一行元素相乘
                c[i * N + j] += tmp * b[k * N + j];
            }
        }
    }
}

// 转置矩阵乘法，a 的一行元素与 b 的一行元素相乘
void matrix_mul_c_v3(int* a, int* b, int* c) {

    int* b_T = (int*)malloc(N * L * sizeof(int));

#if DEBUG == 1
    clock_t start, end;
    start = clock();
#endif

    // 转置 b
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < L; ++j) {
            b_T[i * L + j] = b[j * N + i];
        }
    }

#if DEBUG == 1
	end = clock();
    printf(" v3 transpose time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);
    start = clock();
#endif

    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            int tmp = 0;
            for (int k = 0; k < L; ++k) { // a 的一行元素与 b 的一行元素相乘
                tmp += a[i * N + k] * b_T[j * N + k];
            }
            c[i * N + j] = tmp;
        }
    }

#if DEBUG == 1  
    end = clock();
    printf(" v3 matrix multi time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);
#endif

    free(b_T);
}

/*********************************** cuda model ***********************************/





/*********************************** main ***********************************/
int main()
{
    clock_t start, end;

    int* a = (int*)malloc(M * L * sizeof(int));
    int* b = (int*)malloc(L * N * sizeof(int));
    int* c = (int*)malloc(M * N * sizeof(int));
    
    // init data
    start = clock();
    for (int i = 0; i < M * L; ++i) {
		a[i] = (int)(rand() & 0xFF);
        b[i] = (int)(rand() & 0xFF);
        c[i] = 0;
	}    
    end = clock();
    printf("init time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // matrix multiplication on cpu
    start = clock();
    matrix_mul_c(a, b, c);
	end = clock();
    printf("cpu time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);

    // matrix multiplication on cpu v2
    int* c_v2 = (int*)malloc(M * N * sizeof(int));
    start = clock();
    matrix_mul_c_v2(a, b, c_v2);
	end = clock();
    printf("cpu v2 time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);
    compare_result(c, c_v2);

    // matrix multiplication on cpu v3
    int* c_v3 = (int*)malloc(M * N * sizeof(int));
    start = clock();
    matrix_mul_c_v3(a, b, c_v3);
    end = clock();
    printf("cpu v3 time: %fs\n", (double)(end - start) / CLOCKS_PER_SEC);
    compare_result(c, c_v3);




    // free
    free(a);
    free(b);
    free(c);
    free(c_v2);
    free(c_v3);

    return 0;
}

