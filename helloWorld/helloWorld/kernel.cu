#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void HelloFromGPU(){
	printf("Hello world from GPU.\n");
}

void HelloFromeCPU(void) {
	printf("Hello world from CPU.\n\n");
}

void PrintDeviceProperties(void) {
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Device %d: %s\n", dev, deviceProp.name);
	printf("Number of multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Total amount of shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
	printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
	printf("Warp size: %d\n", deviceProp.warpSize);
	printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Maximum number of warps per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize);
	printf("Maximum grid size: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("Maximum block dimension: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Maximum grid dimension: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("Clock rate: %d MHz\n", deviceProp.clockRate);
	printf("Total constant memory: %lu bytes\n", deviceProp.totalConstMem);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Memory clock rate: %d MHz\n", deviceProp.memoryClockRate);
	printf("Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
	printf("Peak memory bandwidth: %f GB/s\n", 2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8) / 1.0e6);
	printf("Device overlap: %s\n", deviceProp.deviceOverlap ? "Enabled" : "Disabled");
	printf("Kernel execution timeout: %s\n", deviceProp.kernelExecTimeoutEnabled ? "Enabled" : "Disabled");
	printf("Integrated: %s\n", deviceProp.integrated ? ")Yes" : "No");
}

int main(void) {

	HelloFromeCPU();

	HelloFromGPU <<<1, 10 >>> ();

	PrintDeviceProperties();

	return 0;
}

